#include <hip/hip_runtime.h>

__global__ void custom_kernel(float *input, float *output, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    output[idx] = input[idx] * 2.0f; // Example: multiply by 2
  }
}

extern "C" void custom_kernel_launcher(float *input, float *output, int size) {
  int block_size = 256;
  int grid_size = (size + block_size - 1) / block_size;
  custom_kernel<<<grid_size, block_size>>>(input, output, size);
}
